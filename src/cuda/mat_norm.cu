#include "hip/hip_runtime.h"
#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/utility.hpp"
#include <cuda/mat_norm.hpp>
#include <hip/hip_runtime_api.h>
#include <Utils.hpp>

using namespace cv;
using namespace cv::cuda;
using namespace cv::cuda::device;

namespace ORB_SLAM2 { namespace cuda {

MatNormGPU::MatNormGPU() {
    // GpuMat should be already stored in memory
}

MatNormGPU::~MatNormGPU() {
}

__global__
void kernel_get_mat_pixel (uint8_t * src, int w)
{
    // assuming that we resized it to CV_32F so the channel number is 1
    subtract_val = src[ (w*step) + (w)];
}

void MatNormGPU::setSubtractValue(cv::cuda::GpuMat _img, int w)
{
    kernel_get_mat_pixel<<1, 1>>(_img.data, w);
}

__global__
void kernel_subtract_pixel_from_mat (uint8_t * src, int MaxRows, int MaxCols, int step)
{
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x; //Row number
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y; //Column number
    //unsigned int ch = blockIdx.z * blockDim.z + threadIdx.z; //Channel 0

    if (row<MaxRows && col<MaxCols) {
        int idx = row * step + col; // maxChannels is 1 and ch is 0
        src[idx] = src[idx] - subtract_val;
    }
}

void MatNormGPU::subtract_pixel_from_mat (cv::cuda::GpuMat _img)
{
    dim3 tpb(16, 16);
    dim3 bpg(((_img.cols + 15) / 16), ((_img.rows + 15)/ 16));
    kernel_subtract_pixel_from_mat<<bpg, tpb>> (_img.data,_img.rows, _img.cols, _img.step);
}

} }
